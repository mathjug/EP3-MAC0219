// Bruno Armond Braga Nusp: 12542331
// Matheus Sanches Jurgensen Nusp: 12542199
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <png.h>

// Funcao que aplica a matriz de transformacao A
// ao pixel px = (r, g, b)
// (new_r, new_g, new_b)' = A * (r, g, b)'
__host__ __device__ void modify_pixel(png_bytep px, double *A) {
    double r = px[0] / 255.0;
    double g = px[1] / 255.0;
    double b = px[2] / 255.0;

    double new_r = A[0] * r + A[1] * g + A[2] * b;
    double new_g = A[3] * r + A[4] * g + A[5] * b;
    double new_b = A[6] * r + A[7] * g + A[8] * b;

    new_r = fmin(fmax(new_r, 0.0), 1.0);
    new_g = fmin(fmax(new_g, 0.0), 1.0);
    new_b = fmin(fmax(new_b, 0.0), 1.0);

    px[0] = (png_byte) round(new_r * 255.0);
    px[1] = (png_byte) round(new_g * 255.0);
    px[2] = (png_byte) round(new_b * 255.0);
}

// Altera a matiz (hue) de uma imagem sequencialmente
void modify_hue_seq(png_bytep image, int width, int height, double hue_diff) {
    double c = cos(2 * M_PI * hue_diff);
    double s = sin(2 * M_PI * hue_diff);
    double one_third = 1.0 / 3.0;
    double sqrt_third = sqrt(one_third);

    // Matriz A compoe as operacoes de
    // conversao de RGB para HSV, mudanca de hue,
    // e conversao de HSV de volta para RGB
    // (new_r, new_g, new_b)' = A * (r, g, b)'
    // https://stackoverflow.com/questions/8507885/shift-hue-of-an-rgb-color

    double a11 = c + one_third * (1.0 - c);
    double a12 = one_third * (1.0 - c) - sqrt_third * s;
    double a13 = one_third * (1.0 - c) + sqrt_third * s;
    double a21 = a13; double a22 = a11; double a23 = a12;
    double a31 = a12; double a32 = a13; double a33 = a11;

    double A[9] = {a11, a12, a13, a21, a22, a23, a31, a32, a33};

    for (int i = 0; i < height; i++) {
        png_bytep row = &(image[i * width * 3]);
        for (int j = 0; j < width; j++) {
            png_bytep px = &(row[j * 3]);
            modify_pixel(px, A);
        }
    }
}

// Funcao auxiliar para identificar erros CUDA
void checkErrors(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s [Erro CUDA: %s]\n",
                msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void calculates_A(double hue_diff, double *A) {
    double c = cos(2 * M_PI * hue_diff);
    double s = sin(2 * M_PI * hue_diff);
    double one_third = 1.0 / 3.0;
    double sqrt_third = sqrt(one_third);

    double a11 = c + one_third * (1.0 - c);
    double a12 = one_third * (1.0 - c) - sqrt_third * s;
    double a13 = one_third * (1.0 - c) + sqrt_third * s;

    A[0] = a11;
    A[1] = a12;
    A[2] = a13;
    A[4] = A[8] = A[0];
    A[5] = A[6] = A[1];
    A[3] = A[7] = A[2];
}

// Kernel CUDA para alteracao do hue
// Voce deve modificar essa funcao no EP3
__global__ void modify_hue_kernel(png_bytep d_image,
                                  int width,
                                  int height,
                                  double *A) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = (y * width + x) * 3;
        png_bytep px = &(d_image[index]);
        modify_pixel(px, A);
    }
}

// Altera a matiz (hue) de uma imagem em paralelo
void modify_hue(png_bytep h_image,
                int width,
                int height,
                size_t image_size,
                double hue_diff) {

    int size_A = 9 * sizeof(double); // matriz 3x3
    double *h_A = (double *) malloc(size_A); // alocando matriz A no host
    calculates_A(hue_diff, h_A);

    double *d_A;
    hipMalloc(&d_A, size_A);
    checkErrors(hipGetLastError(), "Alocacao da matriz A no device");

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    checkErrors(hipGetLastError(), "Copia da matriz A para o device");

    png_bytep d_image;
    hipMalloc((void**)&d_image, image_size);
    checkErrors(hipGetLastError(), "Alocacao da imagem no device");

    hipMemcpy(d_image, h_image, image_size, hipMemcpyHostToDevice);
    checkErrors(hipGetLastError(), "Copia da imagem para o device");

    // blocos por grid e threads por bloco
    dim3 dim_block(32, 32);
    dim3 dim_grid((width + dim_block.x - 1) / dim_block.x, (height + dim_block.y - 1) / dim_block.y);


    modify_hue_kernel<<<dim_grid, dim_block>>>(d_image, width, height, d_A);
    checkErrors(hipGetLastError(), "Lançamento do kernel");

    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
    checkErrors(hipGetLastError(), "Copia da imagem para o host");

    hipFree(d_image);
    hipFree(d_A);
    free(h_A);
}

// Le imagem png de um arquivo de entrada para a memoria
void read_png_image(const char *filename,
                    png_bytep *image,
                    int *width,
                    int *height,
                    size_t *image_size) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Erro ao ler o arquivo de entrada %s\n", filename);
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fprintf(stderr, "Erro ao criar PNG read struct \n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        fprintf(stderr, "Erro ao criar PNG info struct \n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Em caso de erro nas funcoes da libpng,
    // programa "pula" para este ponto de execucao
    if (setjmp(png_jmpbuf(png))) {
        fprintf(stderr, "Erro ao ler imagem PNG \n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_init_io(png, fp);
    png_read_info(png, info);

    *width = png_get_image_width(png, info);
    *height = png_get_image_height(png, info);
    png_byte color_type = png_get_color_type(png, info);
    png_byte bit_depth = png_get_bit_depth(png, info);

    // Verifica se imagem png possui o formato apropriado
    if ((color_type != PNG_COLOR_TYPE_RGB && color_type != PNG_COLOR_TYPE_GRAY)
        || bit_depth != 8) {
        printf("Formato PNG nao suportado, deve ser 8-bit RGB ou grayscale\n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_read_update_info(png, info);

    // Alocacao de memoria para imagem e ponteiros para as linhas
    *image_size = png_get_rowbytes(png, info) * (*height);
    *image = (png_bytep) malloc(*image_size);

    png_bytep *row_pointers = (png_bytep *) malloc(sizeof(png_bytep) * (*height));
    for (int i = 0; i < *height; i++) {
        row_pointers[i] = *image + i * png_get_rowbytes(png, info);
    }

    // Leitura da imagem para a memoria
    png_read_image(png, row_pointers);

    // Finalizacao da leitura
    png_destroy_read_struct(&png, &info, NULL);
    fclose(fp);
    free(row_pointers);
}

// Escreve imagem png da memoria para um arquivo de saida
void write_png_image(const char *filename,
                     png_bytep image,
                     int width,
                     int height) {
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Erro ao criar o arquivo de saida %s\n", filename);
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fprintf(stderr, "Erro ao criar PNG write struct \n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        fprintf(stderr, "Erro ao criar PNG info struct.\n");
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Em caso de erro nas funcoes da libpng,
    // programa "pula" para este ponto de execucao
    if (setjmp(png_jmpbuf(png))) {
        printf("Erro ao escrever imagem PNG \n");
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        return;
    }

    png_init_io(png, fp);

    // Configura o formato da imagem a ser criada
    png_set_IHDR(
        png, info, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT
    );

    png_write_info(png, info);

    // Criacao de ponteiros para as linhas
    png_bytep row_pointers[height];
    for (int i = 0; i < height; i++) {
        row_pointers[i] = &(image[i * width * 3]);
    }

    // Escrita da imagem a partir da memoria
    png_write_image(png, row_pointers);
    png_write_end(png, NULL);

    // Finalizacao da escrita
    png_destroy_write_struct(&png, &info);
    fclose(fp);
}

int main(int argc, char *argv[]) {
    png_bytep image;
    int width, height;
    size_t image_size;

    // Leitura e validacao dos parametros de entrada
    if (argc != 4) {
        printf("Uso: ./hue_modify <input_file> <output_file> <hue_diff>\n");
        printf("0.0 <= hue_diff <= 1.0\n");
        exit(EXIT_FAILURE);
    }

    double hue_diff;
    int ret = sscanf(argv[3], "%lf", &hue_diff);
    if (ret == 0 || ret == EOF) {
        fprintf(stderr, "Erro ao ler hue_diff\n");
        exit(EXIT_FAILURE);
    }

    if (hue_diff < 0.0 || hue_diff > 1.0) {
        fprintf(stderr, "hue_diff deve ser entre 0.0 e 1.0\n");
        exit(EXIT_FAILURE);
    }

    // Leitura da imagem para memoria
    read_png_image(argv[1], &image, &width, &height, &image_size);

    // Processamento da imagem (alteracao do hue)

    // Versao sequencial:
    //modify_hue_seq(image, width, height, hue_diff);

    // // Versao paralela
    modify_hue(image, width, height, image_size, hue_diff);

    // Escrita da imagem para arquivo
    write_png_image(argv[2], image, width, height);

    // Liberacao de memoria
    free(image);
    return 0;
}

